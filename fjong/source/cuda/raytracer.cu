#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "vec3.h"
#include "ray.h"
#include "marchobject.h"
#include "world.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cout << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}



/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}


/**
 * Host main routine
 */



//__device__ bool raymarchSingle(ray& r, int ignore, int pass, int cnt, World* world, marchobject** objects, int* culled, int tid)//, hitable **world) {
__device__ bool raymarchSingle(ray& r, int ignore, int pass, int cnt, World* world, marchobject** objects)//, hitable **world) {
{
    //marchobject::Init();
     vec3 isp;
     float shadow = 1;
     float t = 1;
     int winner = -1;

     int culled[24];
//     ray culledr[16];


     int len = 0;

     vec3 isp1, isp2;
     float t1, t2;
     for (int i=0;i<world->length;i++) {
         if (len>=24) break;
         if (i!=ignore)
         if (r.IntersectSphere(objects[i]->pos*-1,vec3(1,1,1)*objects[i]->bbRadius,isp1,isp2,t1,t2)) {
             culled[len] = i;
             len++;
         }
     }
     ray rwinner;
     r.curPos = r.org;
     //   world->length=7;
     for (int i=0;i<cnt;i++) {
         float precis = 0.004*t;
         float keep=1000.0;
         //ray.m_curStep =t; //(ray.m_origin-m_objects[j]->m_position).length();
         //ray.setCurrent(t);
         r.curPos = r.point_at_parameter(t);
        // r.point_at_parameter(t);
         int w= -1;
         //            for (int j=0;j<world->length;j++) {//marchobject* ro: culled) {
         for (int l=0;l<len;l++) {//marchobject* ro: culled) {
             int j = culled[l];

             ray rotr = r.Rotate(objects[j]->rotMat,objects[j]->pos);
             float keep2 = objects[j]->intersect(rotr);
             if (keep2<keep) {
                 keep = keep2;
                 w = j;


             }

             if (keep2<precis) {
                 winner = w;
                 i=cnt;
                 if (pass==2) {

                     return true;
                 }
                 rwinner = rotr;
                 break;

             }
         }
         t=t+keep;

     }


     if (winner!=-1) {
            //Ray rotated = winner->m_localRay[tid];//ray.Rotate(winner->m_rotmat, winner->m_position);
             ray rotated = rwinner;
            //ray.m_currentPos = isp;
    //                exit(1);
            isp = rotated.curPos;
            r.intensity = vec3(1,0,0);
            vec3 normal = objects[winner]->CalcMarchNormal(rotated.curPos);
            normal = objects[winner]->rotMatInv*normal;
            vec3 tt(1,2,-213.123);
            vec3 tangent =vec3::cross(tt,normal).normalized();
            vec3 bi = vec3::cross(tangent,normal).normalized();

//            normal  = objects[winner]->GetPerturbedNormal(isp,normal,tangent);


    //        ray.m_reflect = 0;
            vec3 reflectionDir = r.dir-normal*2*dot(r.dir, normal);;
            vec3 lp = r.curPos;//-winner->m_localPos;
     //       ray.m_z=10000;
            objects[winner]->CalculateLight(&r,normal,tangent,lp,world->light0,reflectionDir,objects,0);

//            objects[winner]->reflectivity = 0.9;

            // Reflections
            if (objects[winner]->reflectivity>0 && r.reflect>0) {
                if (objects[winner]->glossiness==1)
                {
                    ray nxt(lp,reflectionDir);
                    nxt.reflect=r.reflect-1;
                    raymarchSingle(nxt, winner, 1, 24,world, objects);
                    r.intensity = r.intensity*(1-objects[winner]->reflectivity) + objects[winner]->reflectivity*nxt.intensity;
                }
                else {
                    // Doesn't work
                    shadow=1;
                    len=0;
                    vec3 in = vec3(0,0,0);
                    for (int j=0;j<shadow;j++) {
                        vec3 disp = vec3(world->rnd[3*j+0]%1024-512,world->rnd[3*j+1]%1024-512,world->rnd[3*j+2]%1024-512).normalized();
                        disp = (disp*3 + reflectionDir.normalized()).normalized();
                        ray nxt(lp,disp);
                        nxt.reflect=0;
                        raymarchSingle(nxt, winner, 1, 24,world, objects);
                        in+=nxt.intensity/(float)shadow;
                    }
                    r.intensity = r.intensity*(1-objects[winner]->reflectivity) + objects[winner]->reflectivity*in;
                   }
                    shadow = 1;
            }
            if (pass==0) {
                ray shadowRay(lp,world->light0);
                if (raymarchSingle(shadowRay, winner, 2,32,world,objects)) {
                    shadow*=0.5;
                }

            }

            r.intensity*=shadow;

            return true;

        }

       world->sky(r);

        return false;
    }

__global__ void create_world(marchobject* objects, marchobject** objectsI, int cnt) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i=0;i<cnt;i++) {
            objectsI[i] = nullptr;
            if (objects[i].type==0) {
                mo_sphere* s = new mo_sphere();
                *(objectsI+i) = s;
                *s =  (mo_sphere&)objects[i];
            }
            if (objects[i].type==1) {
                mo_plane* s = new mo_plane();
                *(objectsI+i) = s;

                *s =  (mo_plane&)objects[i];
//                s->pos = vec3(0,4,0);
            }
            if (objects[i].type==2) {
                mo_box* s = new mo_box();
                *(objectsI+i) = s;
                *s =  (mo_box&)objects[i];
                s->box = objects[i].p2;
//                s->pos = vec3(0,4,0);
            }
            if (objects[i].type==3) {
                mo_torus* s = new mo_torus();
                *(objectsI+i) = s;
                *s = (mo_torus&)objects[i];
//                s->pos = vec3(0,4,0);
            }
            if (objects[i].type==4) {
                mo_cylinder* s = new mo_cylinder();
                *(objectsI+i) = s;
                *s = (mo_cylinder&)objects[i];
//                s->pos = vec3(0,4,0);
            }
        }

/*        *(d_list)   = new sphere(vec3(0,0,-1), 0.5);
        *(d_list+1) = new sphere(vec3(0,-100.5,-1), 100);
        *d_world    = new hitable_list(d_list,2);*/
    }
}

__global__ void delete_world(marchobject* objects, marchobject** objectsI, int cnt) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i=0;i<cnt;i++) {
            if (objectsI[i]!=nullptr)
                delete objectsI[i];
        }
    }

}


__global__ void renderImage(unsigned char *fb, int max_x, int max_y,vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin, World *world, marchobject** objects)
{
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;
   if((i >= max_x) || (j >= max_y)) return;
   int pixel_index = j*max_x + i;
   float u = float(i) / float(max_x);
   float v = float(j) / float(max_y);
   ray r(origin, (lower_left_corner + u*horizontal + v*vertical).normalized());
//   ray r(origin, (u*horizontal + v*vertical).normalized());
   r.reflect = 2;
//   vec3 col(0,0,0);
//   fb[pixel_index] = color(r, world);
   r.intensity = vec3(0,0,0);
   if (raymarchSingle(r,-1,0,90,world, objects)) {
  //     col = r.intensity;
   }
   vec3 in = r.intensity.clamp();
   fb[3*pixel_index] = in.x()*255;
   fb[3*pixel_index+1] = in.y()*255;
   fb[3*pixel_index+2] = in.z()*255;

//   raymarchSingle(const ray& r, int pass, int cnt, World* world)//, hitable **world)
}


/*__global__ void renderImage(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x*3 + i*3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}
*/


unsigned char *fb = nullptr;

unsigned char* RaytraceImage(int nx, int ny, int* img, World* w) {
    int num_pixels = nx*ny;

    size_t fb_size = num_pixels*3;

    // allocate FB
    if (fb==nullptr)
        checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    World* world;
    int bytesw = (sizeof(World));
    int bytesm = (w->length*(sizeof(marchobject)));
    marchobject* objects;
    marchobject** objectsI;
    checkCudaErrors(hipMallocManaged((void **)&world, bytesw));
    checkCudaErrors(hipMallocManaged((void **)&objects, bytesm));
    checkCudaErrors(hipMallocManaged((void **)&objectsI, bytesm));


/*    world->length = w->length;
    for (int i=0;i<w->length;i++)
        world->objects[i] = w->objects[i];*/
    hipMemcpy(world,w,bytesw,hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());


    hipMemcpy(objects, w->objects, bytesm, hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());
    int tx = 8;
    int ty = 8;
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);




    create_world<<<1,1>>>(objects,objectsI, world->length);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    renderImage<<<blocks, threads>>>(fb, nx, ny,
                                    w->lower_left_corner,
                                    w->horizontal,
                                    w->vertical,
                                    w->origin, world, objectsI);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    delete_world<<<1,1>>>(objects,objectsI, world->length);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


       checkCudaErrors(hipFree(objects));
       checkCudaErrors(hipFree(world));

       return fb;

}



void TestCuda()
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");

    system("pause");

}
